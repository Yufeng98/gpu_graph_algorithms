#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <algorithm>
#include <cassert>
#include <iostream>
#include <fstream>
#include <sstream>
#include <vector>
#include <limits>

using std::cout;
using std::endl;

__global__ void relax(int N, int *d_in_V, int *d_in_I, int *d_in_E, int *d_in_W, int *d_out_D, int *d_out_pred) {
    unsigned int tid = threadIdx.x;
    unsigned int index = threadIdx.x + (blockDim.x * blockIdx.x);

    if (index < N - 1) {
        for (int j = d_in_I[index]; j < d_in_I[index + 1]; j++) {
            int u = d_in_V[index];
            //int v = d_in_V[d_in_E[j]];
            int w = d_in_W[j];
            int du = d_out_D[index];
            int dv = d_out_D[d_in_E[j]];
            if (du + w < dv) {
                atomicExch(&d_out_D[d_in_E[j]], du + w);
                atomicExch(&d_out_pred[d_in_E[j]], u);
            }
        }
    }
}

int main (int argc, char **argv) {

    //input
    std::vector<int> V = {1, 2, 3, 4, 5};
    std::vector<int> I = {0, 2, 5, 6, 8, 10};
    //std::vector<int> E = {2, 4, 3, 4, 5, 2, 3, 5, 1, 3}; // This E stores destination vertex for each edge from V[I[i]].. V[I[i+1]]
    std::vector<int> E = {1, 3, 2, 3, 4, 1, 2, 4, 0, 2}; // This E array stores index of destination vertex instead of actual vertex itself. So V[E[i]] is the vertex
    std::vector<int> W = {6, 7, 5, 8, -4, -2, -3, 9, 2, 7};

    //output
    std::vector<int> D(V.size(), std::numeric_limits<int>::max()); //Shortest path of V[i] from source
    std::vector<int> pred(V.size(), -1); // Predecessor vetex of V[i]

    //Set source vertex and predecessor
    D[0] = 0;
    pred[0] = 0;

    //int *in_V = V.data();
    //int *in_I = I.data();
    //int *in_E = E.data();
    //int *in_W = W.data();

    int N = I.size();
    int BLOCKS = 1;
    int BLOCK_SIZE = 16;
    BLOCKS = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

    int dev = 0;
    hipSetDevice(dev);
    hipDeviceProp_t devProps;
    if (hipGetDeviceProperties(&devProps, dev) == 0)
    {
        printf("Using device %d:\n", dev);
        printf("%s; global mem: %dB; compute v%d.%d; clock: %d kHz\n",
               devProps.name, (int)devProps.totalGlobalMem,
               (int)devProps.major, (int)devProps.minor,
               (int)devProps.clockRate);
    }

    cout << "Blocks : " << BLOCKS << " Block size : " << BLOCK_SIZE << endl;

    int *d_in_V;
    int *d_in_I;
    int *d_in_E;
    int *d_in_W;
    int *d_out_D;
    int *d_out_pred;

    //allocate memory
    hipMalloc((void**) &d_in_V, V.size() *sizeof(int));
    hipMalloc((void**) &d_in_I, I.size() *sizeof(int));
    hipMalloc((void**) &d_in_E, E.size() *sizeof(int));
    hipMalloc((void**) &d_in_W, W.size() *sizeof(int));

    hipMalloc((void**) &d_out_D, V.size() *sizeof(int));
    hipMalloc((void**) &d_out_pred, V.size() *sizeof(int));

    //copy to device memory
    hipMemcpy(d_in_V, V.data(), V.size() *sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_in_I, I.data(), I.size() *sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_in_E, E.data(), E.size() *sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_in_W, W.data(), W.size() *sizeof(int), hipMemcpyHostToDevice);

    hipMemcpy(d_out_D, D.data(), D.size() *sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_out_pred, pred.data(), pred.size() *sizeof(int), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Bellman ford
    for (int round = 1; round < V.size(); round++) {
        cout<< "***** round = " << round << " ******* " << endl;
        relax<<<BLOCKS, BLOCK_SIZE>>>(N, d_in_V, d_in_I, d_in_E, d_in_W, d_out_D, d_out_pred);
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    int *out_path = new int[V.size()];
    int *out_pred = new int[V.size()];

    hipMemcpy(out_path, d_out_D, D.size()*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(out_pred, d_out_pred, pred.size()*sizeof(int), hipMemcpyDeviceToHost);

    cout << "Shortest Path : " << endl;
    for (int i = 0; i < D.size(); i++) {
        cout << "from " << V[0] << " to " << V[i] << " = " << out_path[i] << " predecessor = " << out_pred[i] << std::endl;
    }

    cout << "average time elapsed : " << elapsedTime << endl;

    free(out_pred);
    free(out_path);
    hipFree(d_in_V);
    hipFree(d_in_I);
    hipFree(d_in_E);
    hipFree(d_in_W);
    hipFree(d_out_D);
    hipFree(d_out_pred);
}
